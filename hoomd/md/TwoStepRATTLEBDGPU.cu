#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: joaander

#include "TwoStepRATTLEBDGPU.cuh"
#include "hoomd/VectorMath.h"
#include "hoomd/HOOMDMath.h"

#include "hoomd/RandomNumbers.h"
#include "hoomd/RNGIdentifiers.h"
using namespace hoomd;

#include <assert.h>

inline __device__ Scalar maxNorm(Scalar3 vec, Scalar resid)
    {
    Scalar vec_norm = sqrt(dot(vec,vec));
    Scalar abs_resid = fabs(resid);
    if ( vec_norm > abs_resid) return vec_norm;
    else return abs_resid;
    }

/*! \file TwoSteRATTLEBDGPU.cu
    \brief Defines GPU kernel code for Brownian integration on the GPU. Used by TwoStepRATTLEBDGPU.
*/

//! Takes the second half-step forward in the Langevin integration on a group of particles with
/*! \param d_pos array of particle positions and types
    \param d_vel array of particle positions and masses
    \param d_image array of particle images
    \param box simulation box
    \param d_diameter array of particle diameters
    \param d_tag array of particle tags
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param nwork Number of group members to process on this GPU
    \param d_net_force Net force on each particle
    \param d_gamma_r List of per-type gamma_rs (rotational drag coeff.)
    \param d_orientation Device array of orientation quaternion
    \param d_torque Device array of net torque on each particle
    \param d_inertia Device array of moment of inertial of each particle
    \param d_angmom Device array of transformed angular momentum quaternion of each particle (see online documentation)
    \param d_gamma List of per-type gammas
    \param n_types Number of particle types in the simulation
    \param use_lambda If true, gamma = lambda * diameter
    \param lambda Scale factor to convert diameter to lambda (when use_lambda is true)
    \param timestep Current timestep of the simulation
    \param seed User chosen random number seed
    \param T Temperature set point
    \param aniso If set true, the system would go through rigid body updates for its orientation
    \param deltaT Amount of real time to step forward in one time step
    \param D Dimensionality of the system
    \param d_noiseless_t If set true, there will be no translational noise (random force)
    \param d_noiseless_r If set true, there will be no rotational noise (random torque)
    \param offset Offset of this GPU into group indices

    This kernel is implemented in a very similar manner to gpu_nve_step_one_kernel(), see it for design details.

    This kernel must be launched with enough dynamic shared memory per block to read in d_gamma
*/
extern "C" __global__
void gpu_rattle_brownian_step_one_kernel(Scalar4 *d_pos,
                                  Scalar4 *d_vel,
                                  int3 *d_image,
                                  const BoxDim box,
                                  const Scalar *d_diameter,
                                  unsigned int *d_rtag,
                                  unsigned int *d_groupTags,
                                  const unsigned int nwork,
                                  const Scalar4 *d_net_force,
                                  const Scalar3 *d_f_brownian,
                                  const Scalar3 *d_gamma_r,
                                  Scalar4 *d_orientation,
                                  Scalar4 *d_torque,
                                  const Scalar3 *d_inertia,
                                  Scalar4 *d_angmom,
                                  const Scalar *d_gamma,
                                  const unsigned int n_types,
                                  const bool use_lambda,
                                  const Scalar lambda,
                                  const unsigned int timestep,
                                  const unsigned int seed,
                                  const Scalar T,
                                  const bool aniso,
                                  const Scalar deltaT,
                                  unsigned int D,
                                  const bool d_noiseless_t,
                                  const bool d_noiseless_r,
                                  const unsigned int offset)
    {
    extern __shared__ char s_data[];

    Scalar3 *s_gammas_r = (Scalar3 *)s_data;
    Scalar *s_gammas = (Scalar *)(s_gammas_r + n_types);

    if (!use_lambda)
        {
        // read in the gamma (1 dimensional array), stored in s_gammas[0: n_type] (Pythonic convention)
        for (int cur_offset = 0; cur_offset < n_types; cur_offset += blockDim.x)
            {
            if (cur_offset + threadIdx.x < n_types)
                s_gammas[cur_offset + threadIdx.x] = d_gamma[cur_offset + threadIdx.x];
            }
        __syncthreads();
        }

    // read in the gamma_r, stored in s_gammas_r[0: n_type], which is s_gamma_r[0:n_type]

    for (int cur_offset = 0; cur_offset < n_types; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < n_types)
            s_gammas_r[cur_offset + threadIdx.x] = d_gamma_r[cur_offset + threadIdx.x];
        }
    __syncthreads();

    // determine which particle this thread works on (MEM TRANSFER: 4 bytes)
    int local_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (local_idx < nwork)
        {
        const unsigned int group_idx = local_idx + offset;

        // determine the particle to work on
        unsigned int tag = d_groupTags[group_idx];
        unsigned int idx = d_rtag[tag];

        Scalar4 postype = d_pos[idx];
        Scalar4 vel = d_vel[idx];
        Scalar4 net_force = d_net_force[idx];
        Scalar3 brownian_force = d_f_brownian[tag];
        int3 image = d_image[idx];

        // calculate the magnitude of the random force
        Scalar gamma;
        if (use_lambda)
            {
            // determine gamma from diameter
            gamma = lambda*d_diameter[idx];
            }
        else
            {
            // determine gamma from type
            unsigned int typ = __scalar_as_int(postype.w);
            gamma = s_gammas[typ];
            }
        Scalar deltaT_gamma = deltaT/gamma;


        // compute the random force
        RandomGenerator rng(RNGIdentifier::TwoStepBD, seed, tag, timestep);

        
        Scalar dx = (net_force.x + brownian_force.x) * deltaT_gamma;
        Scalar dy = (net_force.y + brownian_force.y) * deltaT_gamma;
        Scalar dz = (net_force.z + brownian_force.z) * deltaT_gamma;

	    postype.x += dx;
	    postype.y += dy;
	    postype.z += dz;
// particles may have been moved slightly outside the box by the above steps, wrap them back into place
        box.wrap(postype, image);

        // draw a new random velocity for particle j
        Scalar mass = vel.w;
        Scalar sigma = fast::sqrt(T/mass);
        NormalDistribution<Scalar> norm(sigma);
        vel.x = norm(rng);
        vel.y = norm(rng);
        vel.z = norm(rng);

        // write out data
        d_pos[idx] = postype;
        d_vel[idx] = vel;
        d_image[idx] = image;

        // rotational random force and orientation quaternion updates
        if (aniso)
            {
            unsigned int type_r = __scalar_as_int(d_pos[idx].w);

            // gamma_r is stored in the second half of s_gammas a.k.a s_gammas_r
            Scalar3 gamma_r = s_gammas_r[type_r];
            if (gamma_r.x > 0 || gamma_r.y > 0 || gamma_r.z > 0)
                {
                vec3<Scalar> p_vec;
                quat<Scalar> q(d_orientation[idx]);
                vec3<Scalar> t(d_torque[idx]);
                vec3<Scalar> I(d_inertia[idx]);

                // check if the shape is degenerate
                bool x_zero, y_zero, z_zero;
                x_zero = (I.x < EPSILON); y_zero = (I.y < EPSILON); z_zero = (I.z < EPSILON);

                Scalar3 sigma_r = make_scalar3(fast::sqrt(Scalar(2.0)*gamma_r.x*T/deltaT),
                                               fast::sqrt(Scalar(2.0)*gamma_r.y*T/deltaT),
                                               fast::sqrt(Scalar(2.0)*gamma_r.z*T/deltaT));
                if (d_noiseless_r)
                    sigma_r = make_scalar3(0,0,0);

                // original Gaussian random torque
                // Gaussian random distribution is preferred in terms of preserving the exact math
                vec3<Scalar> bf_torque;
                bf_torque.x = NormalDistribution<Scalar>(sigma_r.x)(rng);
                bf_torque.y = NormalDistribution<Scalar>(sigma_r.y)(rng);
                bf_torque.z = NormalDistribution<Scalar>(sigma_r.z)(rng);

                if (x_zero) bf_torque.x = 0;
                if (y_zero) bf_torque.y = 0;
                if (z_zero) bf_torque.z = 0;

                // use the damping by gamma_r and rotate back to lab frame
                // For Future Updates: take special care when have anisotropic gamma_r
                bf_torque = rotate(q, bf_torque);
                if (D < 3)
                    {
                    bf_torque.x = 0;
                    bf_torque.y = 0;
                    t.x = 0;
                    t.y = 0;
                    }

                // do the integration for quaternion
                q += Scalar(0.5) * deltaT * ((t + bf_torque) / vec3<Scalar>(gamma_r)) * q ;
                q = q * (Scalar(1.0) / slow::sqrt(norm2(q)));
                d_orientation[idx] = quat_to_scalar4(q);

                // draw a new random ang_mom for particle j in body frame
                p_vec.x = NormalDistribution<Scalar>(fast::sqrt(T * I.x))(rng);
                p_vec.y = NormalDistribution<Scalar>(fast::sqrt(T * I.y))(rng);
                p_vec.z = NormalDistribution<Scalar>(fast::sqrt(T * I.z))(rng);
                if (x_zero) p_vec.x = 0;
                if (y_zero) p_vec.y = 0;
                if (z_zero) p_vec.z = 0;

                // !! Note this ang_mom isn't well-behaving in 2D,
                // !! because may have effective non-zero ang_mom in x,y

                // store ang_mom quaternion
                quat<Scalar> p = Scalar(2.0) * q * p_vec;
                d_angmom[idx] = quat_to_scalar4(p);
                }
            }
        }
    }

/*! \param d_pos array of particle positions and types
    \param d_vel array of particle positions and masses
    \param d_image array of particle images
    \param box simulation box
    \param d_diameter array of particle diameters
    \param d_tag array of particle tags
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Net force on each particle
    \param d_gamma_r List of per-type gamma_rs (rotational drag coeff.)
    \param d_orientation Device array of orientation quaternion
    \param d_torque Device array of net torque on each particle
    \param d_inertia Device array of moment of inertial of each particle
    \param d_angmom Device array of transformed angular momentum quaternion of each particle (see online documentation)
    \param rattle_langevin.g_args Collected arguments for gpu_brownian_step_one_kernel()
    \param aniso If set true, the system would go through rigid body updates for its orientation
    \param deltaT Amount of real time to step forward in one time step
    \param D Dimensionality of the system
    \param d_noiseless_t If set true, there will be no translational noise (random force)
    \param d_noiseless_r If set true, there will be no rotational noise (random torque)

    This is just a driver for gpu_brownian_step_one_kernel(), see it for details.
*/
hipError_t gpu_rattle_brownian_step_one(Scalar4 *d_pos,
                                  Scalar4 *d_vel,
                                  int3 *d_image,
                                  const BoxDim& box,
                                  const Scalar *d_diameter,
                                  const unsigned int *d_rtag,
                                  const unsigned int *d_groupTags,
                                  const unsigned int group_size,
                                  const Scalar4 *d_net_force,
                                  const Scalar3 *d_f_brownian,
                                  const Scalar3 *d_gamma_r,
                                  Scalar4 *d_orientation,
                                  Scalar4 *d_torque,
                                  const Scalar3 *d_inertia,
                                  Scalar4 *d_angmom,
                                  const rattle_bd_step_one_args& rattle_bd_args,
                                  const bool aniso,
                                  const Scalar deltaT,
                                  const unsigned int D,
                                  const bool d_noiseless_t,
                                  const bool d_noiseless_r,
                                  const GPUPartition& gpu_partition
                                  )
    {
    unsigned int run_block_size = 256;

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid( (nwork/run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel
        gpu_rattle_brownian_step_one_kernel<<< grid, threads, (unsigned int)(sizeof(Scalar)*rattle_bd_args.n_types + sizeof(Scalar3)*rattle_bd_args.n_types)>>>
                                    (d_pos,
                                     d_vel,
                                     d_image,
                                     box,
                                     d_diameter,
                                     d_rtag,
                                     d_groupTags,
                                     nwork,
                                     d_net_force,
                                     d_f_brownian,
                                     d_gamma_r,
                                     d_orientation,
                                     d_torque,
                                     d_inertia,
                                     d_angmom,
                                     rattle_bd_args.d_gamma,
                                     rattle_bd_args.n_types,
                                     rattle_bd_args.use_lambda,
                                     rattle_bd_args.lambda,
                                     rattle_bd_args.timestep,
                                     rattle_bd_args.seed,
                                     rattle_bd_args.T,
                                     aniso,
                                     deltaT,
                                     D,
                                     d_noiseless_t,
                                     d_noiseless_r,
                                     range.first);
        }

    return hipSuccess;
    }

extern "C" __global__
void gpu_include_rattle_force_kernel(const Scalar4 *d_pos,
                                  const Scalar4 *d_vel,
                                  const Scalar *d_diameter,
                                  unsigned int *d_rtag,
                                  unsigned int *d_groupTags,
                                  const unsigned int nwork,
                                  Scalar4 *d_net_force,
                                  Scalar4 *d_f_brownian,
                                  Scalar4 *d_net_virial,
                                  const Scalar *d_gamma,
                                  const unsigned int n_types,
                                  const bool use_lambda,
                                  const Scalar lambda,
                                  const unsigned int timestep,
                                  const unsigned int seed,
                                  const Scalar T,
                                  const Scalar eta,
				                  EvaluatorConstraintManifold manifold,
                                  unsigned int net_virial_pitch,
                                  const Scalar deltaT,
                                  const unsigned int offset)
    {
    extern __shared__ char s_data[];

    Scalar3 *s_gammas_r = (Scalar3 *)s_data;
    Scalar *s_gammas = (Scalar *)(s_gammas_r + n_types);

    if (!use_lambda)
        {
        // read in the gamma (1 dimensional array), stored in s_gammas[0: n_type] (Pythonic convention)
        for (int cur_offset = 0; cur_offset < n_types; cur_offset += blockDim.x)
            {
            if (cur_offset + threadIdx.x < n_types)
                s_gammas[cur_offset + threadIdx.x] = d_gamma[cur_offset + threadIdx.x];
            }
        __syncthreads();
        }

    // read in the gamma_r, stored in s_gammas_r[0: n_type], which is s_gamma_r[0:n_type]

    for (int cur_offset = 0; cur_offset < n_types; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < n_types)
            s_gammas_r[cur_offset + threadIdx.x] = d_gamma_r[cur_offset + threadIdx.x];
        }
    __syncthreads();

    // determine which particle this thread works on (MEM TRANSFER: 4 bytes)
    int local_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (local_idx < nwork)
        {
        const unsigned int group_idx = local_idx + offset;

        // determine the particle to work on
        unsigned int tag = d_groupTags[group_idx];
        unsigned int idx = d_rtag[tag];

        Scalar4 postype = d_pos[idx];
        Scalar4 vel = d_vel[idx];
        Scalar4 net_force = d_net_force[idx];
        Scalar3 brownian_force = d_f_brownian[tag];

        Scalar virial0 = d_net_virial[0*net_virial_pitch+idx];
        Scalar virial1 = d_net_virial[1*net_virial_pitch+idx];
        Scalar virial2 = d_net_virial[2*net_virial_pitch+idx];
        Scalar virial3 = d_net_virial[3*net_virial_pitch+idx];
        Scalar virial4 = d_net_virial[4*net_virial_pitch+idx];
        Scalar virial5 = d_net_virial[5*net_virial_pitch+idx];

        // calculate the magnitude of the random force
        Scalar gamma;
        if (use_lambda)
            {
            // determine gamma from diameter
            gamma = lambda*d_diameter[idx];
            }
        else
            {
            // determine gamma from type
            unsigned int typ = __scalar_as_int(postype.w);
            gamma = s_gammas[typ];
            }
        Scalar deltaT_gamma = deltaT/gamma;


        // compute the random force
        RandomGenerator rng(RNGIdentifier::TwoStepBD, seed, ptag, timestep);

        
	    Scalar3 next_pos;
	    next_pos.x = postype.x;
	    next_pos.y = postype.y;
	    next_pos.z = postype.z;
        Scalar3 normal = manifold.evalNormal(next_pos);

        Scalar rx, ry, rz, coeff;

	    if (T > 0)
	        {
	    	UniformDistribution<Scalar> uniform(Scalar(-1), Scalar(1));
	    	rx = uniform(rng);
	    	ry = uniform(rng);
	    	rz = uniform(rng);

	    	Scalar3 proj = normal;
	    	Scalar proj_norm = 1.0/fast::sqrt(proj.x*proj.x+proj.y*proj.y+proj.z*proj.z);
	    	proj.x *= proj_norm;
	    	proj.y *= proj_norm;
	    	proj.z *= proj_norm;

	    	Scalar proj_r = rx*proj.x + ry*proj.y + rz*proj.z;

	    	rx = rx - proj_r*proj.x;
	    	ry = ry - proj_r*proj.y;
	    	rz = rz - proj_r*proj.z;
	    
                    // compute the bd force (the extra factor of 3 is because <rx^2> is 1/3 in the uniform -1,1 distribution
                    // it is not the dimensionality of the system
                    coeff = fast::sqrt(Scalar(6.0)*T/deltaT_gamma);
                    if (d_noiseless_t)
                        coeff = Scalar(0.0);
	        }
	    else
	        {
               	rx = 0;
               	ry = 0;
               	rz = 0;
               	coeff = 0;
	        }


            brownian_force.x = rx*coeff;
            brownian_force.y = ry*coeff;
            brownian_force.z = rz*coeff;

            // update position

	        Scalar mu = 0;

                unsigned int maxiteration = 10;
	        Scalar inv_alpha = -deltaT_gamma;
	        inv_alpha = Scalar(1.0)/inv_alpha;

	        Scalar3 residual;
	        Scalar resid;
	        unsigned int iteration = 0;

	        do
	        {
	            iteration++;
	            residual.x = postype.x - next_pos.x + (net_force.x + brownian_force.x - mu*normal.x) * deltaT_gamma;
	            residual.y = postype.y - next_pos.y + (net_force.y + brownian_force.y - mu*normal.y) * deltaT_gamma;
	            residual.z = postype.z - next_pos.z + (net_force.z + brownian_force.z - mu*normal.z) * deltaT_gamma;
	            resid = manifold.implicit_function(next_pos);

                    Scalar3 next_normal =  manifold.evalNormal(next_pos);


	            Scalar nndotr = dot(next_normal,residual);
	            Scalar nndotn = dot(next_normal,normal);
	            Scalar beta = (resid + nndotr)/nndotn;

                    next_pos.x = next_pos.x - beta*normal.x + residual.x;   
                    next_pos.y = next_pos.y - beta*normal.y + residual.y;   
                    next_pos.z = next_pos.z - beta*normal.z + residual.z;
	            mu = mu - beta*inv_alpha;
	         
	        } while (maxNorm(residual,resid) > eta && iteration < maxiteration );
    
            net_force.x -= mu*normal.x;
            net_force.y -= mu*normal.y;
            net_force.z -= mu*normal.z;

        virial0 -= mu*normal.x*pos.x;
        virial1 -= 0.5*mu*(normal.x*pos.y+normal.y*pos.x);
        virial2 -= 0.5*mu*(normal.x*pos.z+normal.z*pos.x);
        virial3 -= mu*normal.y*pos.y;
        virial4 -= 0.5*mu*(normal.y*pos.z+normal.z*pos.y);
        virial5 -= mu*normal.z*pos.z;

        d_f_brownian[tag] = brownian_force;

        d_net_force[idx] = net_force;
        d_net_virial[0*net_virial_pitch+idx] = virial0;
        d_net_virial[1*net_virial_pitch+idx] = virial1;
        d_net_virial[2*net_virial_pitch+idx] = virial2;
        d_net_virial[3*net_virial_pitch+idx] = virial3;
        d_net_virial[4*net_virial_pitch+idx] = virial4;
        d_net_virial[5*net_virial_pitch+idx] = virial5;

        }
    }

hipError_t gpu_include_rattle_force(const Scalar4 *d_pos,
                                  const Scalar4 *d_vel,
                                  Scalar4 *d_net_force,
                                  Scalar3 *d_f_brownian,
                                  Scalar *d_net_virial,
                                  const Scalar *d_diameter,
                                  const unsigned int *d_rtag,
                                  const unsigned int *d_groupTags,
                                  const unsigned int group_size,
                                  Scalar4 *d_angmom,
                                  const rattle_bd_step_one_args& rattle_bd_args,
			                      EvaluatorConstraintManifold manifold,
                                  unsigned int net_virial_pitch,
                                  const Scalar deltaT,
                                  const GPUPartition& gpu_partition
                                  )
    {
    unsigned int run_block_size = 256;

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid( (nwork/run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel
        gpu_include_rattle_force_kernel<<< grid, threads, (unsigned int)(sizeof(Scalar)*rattle_bd_args.n_types + sizeof(Scalar3)*rattle_bd_args.n_types)>>>
                                    (d_pos,
                                     d_vel,
                                     d_net_force,
                                     d_net_brownian,
                                     d_net_virial,
                                     d_diameter,
                                     d_rtag,
                                     d_groupTags,
                                     nwork,
                                     rattle_bd_args.d_gamma,
                                     rattle_bd_args.n_types,
                                     rattle_bd_args.use_lambda,
                                     rattle_bd_args.lambda,
                                     rattle_bd_args.timestep,
                                     rattle_bd_args.seed,
                                     rattle_bd_args.T,
			                         EvaluatorConstraintManifold manifold,
                                     rattle_langevin_args.eta,
                                     net_virial_pitch,
                                     deltaT,
                                     range.first);
        }

    return hipSuccess;
    }
