#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: joaander

#include "ActiveVicsekForceComputeGPU.cuh"
#include "hoomd/RandomNumbers.h"
#include "EvaluatorConstraintManifold.h"
#include "hoomd/RNGIdentifiers.h"
using namespace hoomd;

#include <assert.h>


//! Kernel for adjusting active force vectors to align parallel to an ellipsoid surface constraint on the GPU
/*! \param group_size number of particles
    \param d_rtag convert global tag to global index
    \param d_groupTags stores list to convert group index to global tag
    \param d_pos particle positions on device
    \param d_f_actVec particle active force unit vector
    \param d_t_actVec particle active force unit vector
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
*/
__global__ void gpu_compute_active_vicsek_force_set_mean_velocity_kernel(const unsigned int group_size,
                                                   unsigned int *d_rtag,
                                                   unsigned int *d_groupTags,
                                                   Scalar3 *d_f_actVec,
                                                   const Scalar3 *d_f_actVec_backup,
                                                   const unsigned int *d_n_neigh,
                                                   const unsigned int *d_nlist,
                                                   const unsigned int *d_head_list,
                                                   EvaluatorConstraintManifold manifold)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;

    unsigned int tag = d_groupTags[group_idx];
    unsigned int idx = d_rtag[tag];

    const unsigned int myHead = d_head_list[idx];
    const unsigned int size = (unsigned int)d_n_neigh[idx];
    
    Scalar3 mean_vel = d_f_actVec_backup[tag];
    for (unsigned int k = 0; k < size; k++)
        {
        // access the index of this neighbor (MEM TRANSFER: 1 scalar)
        unsigned int j = d_nlist[myHead + k];
        mean_vel += d_f_actVec_backup[j];
        }
    mean_vel /= (size+1);

    Scalar new_norm = Scalar(1.0)/slow::sqrt(mean_vel.x*mean_vel.x + mean_vel.y*mean_vel.y + mean_vel.z*mean_vel.z);

    mean_vel *= new_norm;

    d_f_actVec[tag].x = mean_vel.x;
    d_f_actVec[tag].y = mean_vel.y;
    d_f_actVec[tag].z = mean_vel.z;
    }




hipError_t gpu_compute_active_vicsek_force_set_mean_velocity(const unsigned int group_size,
                                           	   unsigned int *d_rtag,
                                          	   unsigned int *d_groupTags,
                                                   Scalar3 *d_f_actVec,
                                                   const Scalar3 *d_f_actVec_backup,
                                                   const unsigned int *d_n_neigh,
                                                   const unsigned int *d_nlist,
                                                   const unsigned int *d_head_list,
                                           	   EvaluatorConstraintManifold manifold,
                                                   unsigned int block_size)
    {
    // setup the grid to run the kernel
    dim3 grid( group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_compute_active_vicsek_force_set_mean_velocity_kernel<<< grid, threads>>>(group_size,
                                                                    d_rtag,
                                                                    d_groupTags,
                                                                    d_f_actVec,
                                                                    d_f_actVec_backup,
                                                                    d_n_neigh,
                                                                    d_nlist,
                                                                    d_head_list,
                                                                    manifold);
    return hipSuccess;
    }

