#include "hip/hip_runtime.h"
#include "SATSolver.cuh"

#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <hipcub/hipcub.hpp>
#include <cub/iterator/discard_output_iterator.cuh>

#include "hoomd/extern/ECL.cuh"

namespace hpmc {

namespace gpu {

const unsigned int SAT_sentinel = 0xffffffff;

namespace kernel {

__device__ inline bool update_watchlist(
    const unsigned int false_literal,
    unsigned int *d_watch,
    unsigned int *d_next_clause,
    const unsigned int *d_n_clause,
    const unsigned int *d_clause,
    const unsigned int maxn_clause,
    const unsigned int *d_assignment,
    unsigned int *d_next,
    unsigned int &h,
    unsigned int &t)
    {
    unsigned int c = d_watch[false_literal];

    // false_literal is no longer being watched
    d_watch[false_literal] = SAT_sentinel;

    #if 1
    // update the clauses watching it to a different watched literal
    while (c != SAT_sentinel)
        {
        unsigned int next = d_next_clause[c];
        unsigned int n_clause = d_n_clause[c];

        bool found_alternative = false;
        for (unsigned int j = 0; j < n_clause; ++j)
            {
            unsigned int alternative = d_clause[c*maxn_clause+j];
            unsigned int v = alternative >> 1;
            unsigned int a = alternative & 1;
            if (d_assignment[v] == SAT_sentinel || d_assignment[v] == a ^ 1)
                {
                found_alternative = true;

                #if 1
                // the variable corresponding to 'alternative' might become active at this point,
                // because it might not be watched anywhere else. In such a case, we insert it at the
                // 'beginning' of the active ring (that is, just after t)
                if (d_assignment[v] == SAT_sentinel && d_watch[v << 1] == SAT_sentinel && d_watch[(v << 1) | 1] == SAT_sentinel)
                    {
                    if (t == SAT_sentinel)
                        {
                        t = h = v;
                        d_next[t] = h;
                        }
                    else
                        {
                        d_next[v] = h;
                        h = v;
                        d_next[t] = h;
                        }
                    }
                #endif

                // insert clause at begining of alternative literal's watch list
                d_next_clause[c] = d_watch[alternative];
                d_watch[alternative] = c;
                break;
                }
            }

        if (!found_alternative)
            return false; // should never get here

        c = next;
        }
    #endif

    return true;
    }

// Returns true if literal is being watched by a unit clause
__device__ inline bool is_unit(
    const unsigned int literal,
    const unsigned int *d_watch,
    const unsigned int *d_next_clause,
    const unsigned int maxn_clause,
    const unsigned int *d_clause,
    const unsigned int *d_n_clause,
    const unsigned int *d_assignment)
    {
    unsigned int c = d_watch[literal];

    while (c != SAT_sentinel)
        {
        unsigned int n_clause = d_n_clause[c];

        bool unit_clause = true;
        for (unsigned int j = 0; j < n_clause; ++j)
            {
            unsigned int l = d_clause[c*maxn_clause+j];
            if (l == literal)
                continue;

            unsigned int v = l >> 1;
            unsigned int a = l & 1;

            // if there is a different literal that is either unassigned or true, this clause can not be a unit clause
            if (d_assignment[v] == SAT_sentinel || d_assignment[v] == a ^ 1)
                {
                unit_clause = false;
                break;
                }
            }

        if (unit_clause)
            return true;

        c = d_next_clause[c];
        }

    return false;
    }

__global__ void solve_sat(
    unsigned int *d_watch,
    unsigned int *d_next_clause,
    unsigned int *d_next,
    unsigned int *d_h,
    const unsigned int *d_head,
    const unsigned int maxn_clause,
    const unsigned int *d_clause,
    const unsigned int *d_n_clause,
    unsigned int *d_assignment,
    unsigned int *d_state,
    const unsigned int *d_representative,
    const unsigned int n_variables,
    unsigned int *d_unsat,
    unsigned int *d_heap)
    {
    unsigned int node_idx = threadIdx.x + blockIdx.x*blockDim.x;

    if (node_idx >= n_variables)
        return;

    // start from the representatives of every component, all other threads just exit
    if (d_representative[node_idx] != node_idx)
        {
        return;
        }

    unsigned int h = d_head[node_idx];

    // chase pointers until we find a tail for the ring buffer
    unsigned int v = h;
    unsigned int n = 0;
    unsigned int t = SAT_sentinel;
    while (v != SAT_sentinel)
        {
        t = v;
        v = d_next[v];
        n++; // the size of the component
        }
    if (t != SAT_sentinel)
        d_next[t] = h;

    // allocate scratch memory for this component
    unsigned int component_start = atomicAdd(d_heap, n);
    unsigned int d = component_start;

    while (true)
        {
        if (t == SAT_sentinel)
            return; // SAT

        // fetch next variable
        unsigned int k = t;

        bool backtrack = false;
        bool unit = false;
        do
            {
            // look for unit clauses
            h = d_next[k];

            bool is_h_unit = is_unit(h << 1,
                                     d_watch,
                                     d_next_clause,
                                     maxn_clause,
                                     d_clause,
                                     d_n_clause,
                                     d_assignment);
            bool is_neg_h_unit = is_unit((h << 1) | 1,
                                     d_watch,
                                     d_next_clause,
                                     maxn_clause,
                                     d_clause,
                                     d_n_clause,
                                     d_assignment);

            unsigned int f = is_h_unit + (is_neg_h_unit << 1);

            if (f == 1 || f == 2)
                {
                // one of the two literals is true
                d_state[d] = f + 3;
                t = k;
                unit = true;
                break;
                }
            else if (f == 3)
                {
                // conflict
                backtrack = true;
                break;
                }

            k = h;
            }
        while (h != t);

        if (!backtrack && !unit)
            {
            // two way branch
            h = d_next[t];
            d_state[d] = (d_watch[h << 1] == SAT_sentinel) ||
                         (d_watch[(h << 1) | 1] != SAT_sentinel);
            }

        if (!backtrack)
            {
            // move on
            d_h[d++] = k = h;

            if (t == k)
                {
                t = SAT_sentinel;
                }
            else
                {
                // delete k from ring
                d_next[t] = h = d_next[k];
                }
            }
        else
            {
            t = k;

            while (d > component_start && d_state[d-1] >= 2)
                {
                k = d_h[d-1];
                d_assignment[k] = SAT_sentinel;
                if (d_watch[k << 1] != SAT_sentinel || d_watch[(k << 1) | 1] != SAT_sentinel)
                    {
                    d_next[k] = h;
                    h = k;
                    d_next[t] = h;
                    }

                d--;
                }

            if (d == component_start)
                {
                // can't backtrack further, no solutions
                atomicAdd(d_unsat, 1);
                return;
                }
            else
                {
                // backtrack
                d_state[d-1] = 3 - d_state[d-1];
                k = d_h[d-1];
                }
            }

        // update watches
        unsigned int b = (d_state[d-1] + 1) & 1;
        d_assignment[k] = b;
        update_watchlist((k << 1) | b,
                         d_watch,
                         d_next_clause,
                         d_n_clause,
                         d_clause,
                         maxn_clause,
                         d_assignment,
                         d_next,
                         h,
                         t);
        }
    }

__global__ void setup_watch_list(
    unsigned int n_clauses,
    const unsigned int maxn_clause,
    const unsigned int *d_clause,
    const unsigned int *d_n_clause,
    unsigned int *d_watch,
    unsigned int *d_next_clause)
    {
    unsigned int tidx = threadIdx.x + blockDim.x*blockIdx.x;

    if (tidx >= n_clauses)
        return;

    // ignore empty clauses (Is this really necessary and shouldn't the disjunction be false then?)
    if (d_n_clause[tidx] == 0)
        return;

    unsigned int first_literal = d_clause[tidx*maxn_clause];

    // append to the singly linked list for this literal
    unsigned int p = atomicCAS(&d_watch[first_literal], SAT_sentinel, tidx);
    while (p != SAT_sentinel)
        {
        p = atomicCAS(&d_next_clause[p], SAT_sentinel, tidx);
        }
    }

// Initialize the active list for every component.
__global__ void initialize_components(
    unsigned int *d_watch,
    unsigned int *d_assignment,
    const unsigned int *d_component_ptr,
    const unsigned int n_variables,
    unsigned int *d_representative,
    unsigned int *d_head,
    unsigned int *d_next)
    {
    unsigned int node_idx = threadIdx.x + blockIdx.x*blockDim.x;

    if (node_idx >= n_variables)
        return;

    // jump to the node with the lowest index in this component, which is its label
    unsigned int next, vstat = d_component_ptr[node_idx];
    while (vstat > (next = d_component_ptr[vstat]))
        {
        vstat = next;
        }
    unsigned int component = vstat;

    // store the reprentative for this node's component in global mem
    d_representative[node_idx] = component;

    // assign a sentinel value to the variable for this node
    d_assignment[node_idx] = SAT_sentinel;

    if (d_watch[node_idx << 1] != SAT_sentinel || d_watch[(node_idx << 1) | 1] != SAT_sentinel)
        {
        // append ourselves to the linked list
        unsigned int p = atomicCAS(&d_head[component], SAT_sentinel, node_idx);
        while (p != SAT_sentinel)
            {
            p = atomicCAS(&d_next[p], SAT_sentinel, node_idx);
            }
        }
    }

__global__ void parse_clauses(
    const unsigned int n_clauses,
    const unsigned int *d_n_clause,
    const unsigned int *d_clause,
    const unsigned int maxn_clause,
    unsigned int *d_row_counters,
    unsigned int *d_row_head,
    unsigned int *d_row_next)
    {
    const unsigned int tidx = threadIdx.x + blockIdx.x*blockDim.x;

    if (tidx >= n_clauses)
        return;

    unsigned int nclause = d_n_clause[tidx];

    for (unsigned int i = 0; i < nclause; ++i)
        {
        unsigned int l = d_clause[tidx*maxn_clause+i];
        unsigned int v = l >> 1;

        for (unsigned int j = 0; j < nclause; ++j)
            {
            if (j == i)
                continue;

            // increment the row size
            atomicAdd(&d_row_counters[v], 1);

            // link the clause index for this index pair to the row
            unsigned int cidx = tidx*maxn_clause*maxn_clause+j*maxn_clause+i;
            unsigned int p = atomicCAS(&d_row_head[v], SAT_sentinel, cidx);
            while (p != SAT_sentinel)
                {
                p = atomicCAS(&d_row_next[p], SAT_sentinel, cidx);
                }
            }
        }
    }

__global__ void flatten_rows(
    const unsigned int maxn_clause,
    const unsigned int *d_clause,
    const unsigned int nrows,
    const unsigned int *d_row_offset,
    const unsigned int *d_row_head,
    const unsigned int *d_row_next,
    unsigned int *d_colidx)
    {
    const unsigned int tidx = threadIdx.x + blockIdx.x*blockDim.x;

    if (tidx >= nrows)
        return;

    // insert the colum indices into the output array starting at offset, which is a prefix sum
    unsigned int i = d_row_offset[tidx];

    unsigned int p = d_row_head[tidx];
    while (p != SAT_sentinel)
        {
        // load the associated variable
        unsigned int v = d_clause[p / maxn_clause] >> 1;
        d_colidx[i++] = v;

        // look up the next column
        p = d_row_next[p];
        }
    }

} //end namespace kernel

void identify_connected_components(
    const unsigned int n_clauses,
    const unsigned int maxn_clause,
    const unsigned int *d_clause,
    const unsigned int *d_n_clause,
    unsigned int *d_row_counters,
    unsigned int *d_row_head,
    unsigned int *d_row_next,
    unsigned int *d_row_offset,
    unsigned int *d_colidx,
    const unsigned int n_variables,
    unsigned int *d_component_ptr,
    unsigned int *d_work,
    const hipDeviceProp_t devprop,
    const unsigned int block_size,
    CachedAllocator &alloc)
    {
    // set sentinel values
    hipMemsetAsync(d_row_head, 0xff, sizeof(unsigned int)*n_variables);
    hipMemsetAsync(d_row_next, 0xff, sizeof(unsigned int)*n_clauses*maxn_clause*maxn_clause);

    // initialize counter
    hipMemsetAsync(d_row_counters, 0, sizeof(unsigned int)*(n_variables+1));

    // go through the clauses and find dependencies between variables
    hipLaunchKernelGGL(kernel::parse_clauses, n_clauses/block_size + 1, block_size, 0, 0,
        n_clauses,
        d_n_clause,
        d_clause,
        maxn_clause,
        d_row_counters,
        d_row_head,
        d_row_next);

    // prefix sum over row counts
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage,
        temp_storage_bytes,
        d_row_counters,
        d_row_offset,
        n_variables+1);
    d_temp_storage = alloc.allocate(temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage,
        temp_storage_bytes,
        d_row_counters,
        d_row_offset,
        n_variables+1);
    alloc.deallocate((char *)d_temp_storage);

    // flatten into CSR format
    hipLaunchKernelGGL(kernel::flatten_rows, n_variables/block_size + 1, block_size, 0, 0,
        maxn_clause,
        d_clause,
        n_variables,
        d_row_offset,
        d_row_head,
        d_row_next,
        d_colidx);

    unsigned int nnz;
    hipMemcpy(&nnz, &d_row_offset[n_variables], sizeof(unsigned int), hipMemcpyDeviceToHost);

    // find connected components
    ecl_connected_components(
        n_variables,
        nnz,
        (const int *) d_row_offset,
        (const int *) d_colidx,
        (int *) d_component_ptr,
        (int *) d_work,
        devprop,
        false);
    }

// solve the satisfiability problem
void solve_sat(unsigned int *d_watch,
    unsigned int *d_next_clause,
    unsigned int *d_head,
    unsigned int *d_next,
    unsigned int *d_h,
    unsigned int *d_state,
    const unsigned int maxn_clause,
    const unsigned int *d_clause,
    const unsigned int *d_n_clause,
    unsigned int *d_assignment,
    const unsigned int n_variables,
    const unsigned int n_clauses,
    unsigned int *d_unsat,
    const unsigned int *d_component_ptr,
    unsigned int *d_representative,
    unsigned int *d_heap,
    const unsigned int block_size)
    {
    hipMemsetAsync(d_unsat, 0, sizeof(unsigned int));
    hipMemsetAsync(d_heap, 0, sizeof(unsigned int));

    // initialize with sentinel values
    hipMemsetAsync(d_head, 0xff, sizeof(unsigned int)*n_variables);
    hipMemsetAsync(d_next, 0xff, sizeof(unsigned int)*n_variables);
    hipMemsetAsync(d_watch, 0xff, sizeof(unsigned int)*2*n_variables);
    hipMemsetAsync(d_next_clause, 0xff, sizeof(unsigned int)*n_clauses);

    hipLaunchKernelGGL(kernel::setup_watch_list, n_clauses/block_size + 1, block_size, 0, 0,
        n_clauses,
        maxn_clause,
        d_clause,
        d_n_clause,
        d_watch,
        d_next_clause);

    unsigned int sat_block_size = 256;
    hipLaunchKernelGGL(kernel::initialize_components, n_variables/sat_block_size + 1, sat_block_size, 0, 0,
        d_watch,
        d_assignment,
        d_component_ptr,
        n_variables,
        d_representative,
        d_head,
        d_next);

    hipLaunchKernelGGL(kernel::solve_sat, n_variables/sat_block_size + 1, sat_block_size, 0, 0,
        d_watch,
        d_next_clause,
        d_next,
        d_h,
        d_head,
        maxn_clause,
        d_clause,
        d_n_clause,
        d_assignment,
        d_state,
        d_representative,
        n_variables,
        d_unsat,
        d_heap);
    }

} //end namespace gpu
} //end namespace hpm

#undef check_cusparse
