#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "IntegratorHPMCMonoGPUTypes.cuh"
#include "hoomd/GPUPartition.cuh"
#include "hoomd/RandomNumbers.h"
#include "hoomd/RNGIdentifiers.h"

namespace hpmc
{
namespace gpu
{
namespace kernel
{

//! Kernel to generate expanded cells
/*! \param d_excell_idx Output array to list the particle indices in the expanded cells
    \param d_excell_size Output array to list the number of particles in each expanded cell
    \param excli Indexer for the expanded cells
    \param d_cell_idx Particle indices in the normal cells
    \param d_cell_size Number of particles in each cell
    \param d_cell_adj Cell adjacency list
    \param ci Cell indexer
    \param cli Cell list indexer
    \param cadji Cell adjacency indexer
    \param ngpu Number of active devices

    gpu_hpmc_excell_kernel executes one thread per cell. It gathers the particle indices from all neighboring cells
    into the output expanded cell.
*/
__global__ void hpmc_excell(unsigned int *d_excell_idx,
                            unsigned int *d_excell_size,
                            const Index2D excli,
                            const unsigned int *d_cell_idx,
                            const unsigned int *d_cell_size,
                            const unsigned int *d_cell_adj,
                            const Index3D ci,
                            const Index2D cli,
                            const Index2D cadji,
                            const unsigned int ngpu)
    {
    // compute the output cell
    unsigned int my_cell = 0;
    my_cell = blockDim.x * blockIdx.x + threadIdx.x;

    if (my_cell >= ci.getNumElements())
        return;

    unsigned int my_cell_size = 0;

    // loop over neighboring cells and build up the expanded cell list
    for (unsigned int offset = 0; offset < cadji.getW(); offset++)
        {
        unsigned int neigh_cell = d_cell_adj[cadji(offset, my_cell)];

        // iterate over per-device cell lists
        for (unsigned int igpu = 0; igpu < ngpu; ++igpu)
            {
            unsigned int neigh_cell_size = d_cell_size[neigh_cell+igpu*ci.getNumElements()];

            for (unsigned int k = 0; k < neigh_cell_size; k++)
                {
                // read in the index of the new particle to add to our cell
                unsigned int new_idx = d_cell_idx[cli(k, neigh_cell)+igpu*cli.getNumElements()];
                d_excell_idx[excli(my_cell_size, my_cell)] = new_idx;
                my_cell_size++;
                }
            }
        }

    // write out the final size
    d_excell_size[my_cell] = my_cell_size;
    }

//! Kernel for grid shift
/*! \param d_postype postype of each particle
    \param d_image Image flags for each particle
    \param N number of particles
    \param box Simulation box
    \param shift Vector by which to translate the particles

    Shift all the particles by a given vector.

    \ingroup hpmc_kernels
*/
__global__ void hpmc_shift(Scalar4 *d_postype,
                          int3 *d_image,
                          const unsigned int N,
                          const BoxDim box,
                          const Scalar3 shift)
    {
    // identify the active cell that this thread handles
    unsigned int my_pidx = blockIdx.x * blockDim.x + threadIdx.x;

    // this thread is inactive if it indexes past the end of the particle list
    if (my_pidx >= N)
        return;

    // pull in the current position
    Scalar4 postype = d_postype[my_pidx];

    // shift the position
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    pos += shift;

    // wrap the particle back into the box
    int3 image = d_image[my_pidx];
    box.wrap(pos, image);

    // write out the new position and orientation
    d_postype[my_pidx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
    d_image[my_pidx] = image;
    }

//!< Kernel to accept/reject
__global__ void hpmc_sum_energies(const unsigned int *d_update_order_by_ptl,
                 const unsigned int *d_trial_move_type,
                 const Scalar4 *d_trial_vel,
                 const unsigned int *d_reject_out_of_cell,
                 unsigned int *d_reject,
                 unsigned int *d_reject_out,
                 const unsigned int *d_nneigh,
                 const unsigned int *d_nlist,
                 const unsigned int N_old,
                 const unsigned int N,
                 const unsigned int nwork,
                 const unsigned work_offset,
                 const unsigned int maxn,
                 bool patch,
                 const unsigned int *d_nlist_patch_old,
                 const unsigned int *d_nlist_patch_new,
                 const unsigned int *d_nneigh_patch_old,
                 const unsigned int *d_nneigh_patch_new,
                 const float *d_energy_old,
                 const float *d_energy_new,
                 const unsigned int maxn_patch,
                 const unsigned int *d_deltaF_or_nneigh,
                 const unsigned int *d_deltaF_or_len,
                 const unsigned int *d_deltaF_or_nlist,
                 const float *d_deltaF_or_energy,
                 const Scalar *d_deltaF_or,
                 const unsigned int *d_deltaF_or_config,
                 const unsigned maxn_deltaF_or,
                 const unsigned int *d_deltaF_nor_nneigh,
                 const unsigned int *d_deltaF_nor_len,
                 const unsigned int *d_deltaF_nor_k,
                 const unsigned int *d_deltaF_nor_nlist,
                 const float *d_deltaF_nor_energy,
                 const Scalar *d_deltaF_nor,
                 Scalar *d_F,
                 const unsigned int maxn_deltaF_nor,
                 const bool have_auxiliary_variables,
                 const unsigned int max_neighbors,
                 unsigned int *d_req_neighbors
                 )
    {
    unsigned offset = threadIdx.x;
    unsigned int group_size = blockDim.x;
    unsigned int group = threadIdx.y;
    unsigned int n_groups = blockDim.y;
    bool master = offset == 0;

    // the particle we are handling
    unsigned int i = blockIdx.x*n_groups + group;
    bool active = true;
    if (i >= nwork)
        active = false;
    i += work_offset;

    extern __shared__ char sdata[];

    float *s_energy_old = (float *) sdata;
    float *s_energy_new = (float *) (s_energy_old + n_groups);
    float *s_deltaF = (float *) (s_energy_new + n_groups);
    unsigned int *s_reject = (unsigned int *) (s_deltaF + n_groups);

    unsigned int *s_lookup_old = (unsigned int *) (s_reject + n_groups);
    unsigned int *s_sign_old = (unsigned int *) (s_lookup_old + n_groups*max_neighbors);
    unsigned int *s_lookup_new = (unsigned int *) (s_sign_old + n_groups*max_neighbors);
    unsigned int *s_sign_new = (unsigned int *) (s_lookup_new + n_groups*max_neighbors);
    unsigned int *s_len_old = (unsigned int *) (s_sign_new + n_groups*max_neighbors);
    unsigned int *s_len_new = (unsigned int *) (s_len_old + n_groups);

    bool move_active = false;
    if (active && master)
        {
        s_reject[group] = d_reject_out_of_cell[i];
        s_energy_old[group] = 0.0f;
        s_energy_new[group] = 0.0f;
        s_deltaF[group] = 0.0f;

        s_len_old[group] = 0;
        s_len_new[group] = 0;
        }

    if (active)
        {
        move_active = d_trial_move_type[i] > 0;
        }

    __syncthreads();

    if (active && move_active)
        {
        unsigned int update_order_i = d_update_order_by_ptl[i];

        // iterate over overlapping neighbors in old configuration
        unsigned int nneigh = d_nneigh[i];
        bool accept = true;
        for (unsigned int cur_neigh = offset; cur_neigh < nneigh; cur_neigh += group_size)
            {
            unsigned int primitive = d_nlist[cur_neigh+maxn*i];

            unsigned int j = primitive;
            bool old = true;
            if (j >= N_old)
                {
                j -= N_old;
                old = false;
                }

            // has j been updated? ghost particles are not updated
            bool j_has_been_updated = j < N && d_trial_move_type[j]
                && d_update_order_by_ptl[j] < update_order_i && !d_reject[j];

            // acceptance, reject if current configuration of particle overlaps
            if ((old && !j_has_been_updated) || (!old && j_has_been_updated))
                {
                accept = false;
                break;
                }

            } // end loop over neighbors

        if (!accept)
            {
            atomicMax(&s_reject[group], 1);
            }

        if (patch)
            {
            // iterate over overlapping neighbors in old configuration
            float energy_old = 0.0f;
            unsigned int nneigh = d_nneigh_patch_old[i];
            bool evaluated = false;
            for (unsigned int cur_neigh = offset; cur_neigh < nneigh; cur_neigh += group_size)
                {
                unsigned int primitive = d_nlist_patch_old[cur_neigh+maxn_patch*i];

                unsigned int j = primitive;
                bool old = true;
                if (j >= N_old)
                    {
                    j -= N_old;
                    old = false;
                    }

                // has j been updated? ghost particles are not updated
                bool j_has_been_updated = j < N && d_trial_move_type[j]
                    && d_update_order_by_ptl[j] < update_order_i && !d_reject[j];

                if ((old && !j_has_been_updated) || (!old && j_has_been_updated))
                    {
                    energy_old += d_energy_old[cur_neigh+maxn_patch*i];
                    evaluated = true;
                    }

                } // end loop over neighbors

            if (evaluated)
                atomicAdd(&s_energy_old[group], energy_old);

            // iterate over overlapping neighbors in new configuration
            float energy_new = 0.0f;
            nneigh = d_nneigh_patch_new[i];
            evaluated = false;
            for (unsigned int cur_neigh = offset; cur_neigh < nneigh; cur_neigh += group_size)
                {
                unsigned int primitive = d_nlist_patch_new[cur_neigh+maxn_patch*i];

                unsigned int j = primitive;
                bool old = true;
                if (j >= N_old)
                    {
                    j -= N_old;
                    old = false;
                    }

                // has j been updated? ghost particles are not updated
                bool j_has_been_updated = j < N && d_trial_move_type[j]
                    && d_update_order_by_ptl[j] < update_order_i && !d_reject[j];

                if ((old && !j_has_been_updated) || (!old && j_has_been_updated))
                    {
                    energy_new += d_energy_new[cur_neigh+maxn_patch*i];
                    evaluated = true;
                    }

                } // end loop over neighbors

            if (evaluated)
                atomicAdd(&s_energy_new[group], energy_new);
            }

        if (have_auxiliary_variables && master)
            {
            // depletants with auxiliary ntrial != 0
            unsigned int nneigh = d_deltaF_or_nneigh[i];
            unsigned int nterms = 0;
            unsigned int sign_i = 0;
            for (unsigned int cur_neigh = 0; cur_neigh < nneigh; cur_neigh += nterms)
                {
                nterms = d_deltaF_or_len[maxn_deltaF_or*i + cur_neigh];
                bool new_config = d_deltaF_or_config[maxn_deltaF_or*i + cur_neigh];

                bool has_overlap = false;
                float U_j = 0.0;
                for (unsigned int cur_term = cur_neigh; cur_term < cur_neigh + nterms; ++cur_term)
                    {
                    unsigned int j_flag = d_deltaF_or_nlist[maxn_deltaF_or*i + cur_term];

                    unsigned int j = j_flag >> 2;
                    bool old = j_flag & 2;

                    // has j been updated? ghost particles are not updated
                    bool j_has_been_updated = j < N && d_trial_move_type[j]
                        && d_update_order_by_ptl[j] < update_order_i && !d_reject[j];

                    if ((old && !j_has_been_updated) || (!old && j_has_been_updated))
                        {
                        if (j_flag & 1)
                            {
                            // shortcut
                            has_overlap = true;
                            break;
                            }
                        U_j += d_deltaF_or_energy[maxn_deltaF_or*i + cur_term];
                        }
                    }

                float f_j = has_overlap + (1-has_overlap)*(1.0f-fast::exp(-U_j));
                if (f_j != 0.0)
                    {
                    float f_i = d_deltaF_or[maxn_deltaF_or*i + cur_neigh];
                    if (new_config)
                        atomicAdd(&s_deltaF[group], logf(fabsf(1+f_i*f_j)));
                    else
                        atomicAdd(&s_deltaF[group], -logf(fabsf(1+f_i*f_j)));

                    if (new_config && f_i*f_j < -1)
                        sign_i ^= 1;
                    }
                } // end loop over terms

            if (sign_i != __scalar_as_int(d_trial_vel[i].y))
                {
                atomicMax(&s_reject[group], 1);
                }

            nneigh = d_deltaF_nor_nneigh[i];
            nterms = 0;
            for (unsigned int cur_neigh = 0; cur_neigh < nneigh; cur_neigh += nterms)
                {
                nterms = d_deltaF_nor_len[maxn_deltaF_nor*i + cur_neigh];
                unsigned int k_flag = d_deltaF_nor_k[maxn_deltaF_nor*i + cur_neigh];

                unsigned int k = k_flag >> 2;
                bool k_old = k_flag & 2;
                bool i_old = k_flag & 1;

                // has the inserting particle k been updated? ghost particles are not updated
                bool k_has_been_updated = k < N && d_trial_move_type[k]
                    && d_update_order_by_ptl[k] < update_order_i && !d_reject[k];

                if ((k_old && k_has_been_updated) || (!k_old && !k_has_been_updated))
                    continue;

                bool has_overlap = false;
                bool has_overlap_other = false;
                float U_j = 0.0;
                float U_j_other = 0.0;
                for (unsigned int cur_term = cur_neigh; cur_term < cur_neigh + nterms; ++cur_term)
                    {
                    unsigned int j_flag = d_deltaF_nor_nlist[maxn_deltaF_nor*i + cur_term];

                    unsigned int j = j_flag >> 2;
                    bool j_old = j_flag & 2;

                    if (j == i && i_old != j_old)
                        {
                        continue;
                        }

                    // has j been updated? ghost particles are not updated
                    bool j_has_been_updated = j < N && d_trial_move_type[j]
                        && d_update_order_by_ptl[j] < update_order_i && !d_reject[j];

                    if (i == j || (j_old && !j_has_been_updated) || (!j_old && j_has_been_updated))
                        {
                        if (j_flag & 1)
                            {
                            has_overlap = true;

                            if (j != i)
                                has_overlap_other = true;
                            }

                        float t = d_deltaF_nor_energy[maxn_deltaF_nor*i + cur_term];
                        U_j += t;
                        if (j != i)
                            U_j_other += t;
                        }
                    }

                float f_j = has_overlap + (1-has_overlap)*(1.0f-fast::exp(-U_j));
                float f_k = d_deltaF_nor[maxn_deltaF_nor*i + cur_neigh];
                unsigned int sign_k = 0;
                if (f_j != 0.0)
                    {
                    if (!i_old)
                        atomicAdd(&s_deltaF[group], logf(fabs(1+f_k*f_j)));
                    else
                        atomicAdd(&s_deltaF[group], -logf(fabs(1+f_k*f_j)));

                    if (f_k*f_j < -1.0f)
                        sign_k ^= 1;
                    }

                // add back neighbor term (excluding i) on the other side of the fraction
                float f_j_other = has_overlap_other + (1-has_overlap_other)*(1.0f-fast::exp(-U_j_other));
                if (f_j_other != 0.0)
                    {
                    if (!i_old)
                        atomicAdd(&s_deltaF[group], -logf(fabs(1+f_k*f_j_other)));
                    else
                        atomicAdd(&s_deltaF[group], logf(fabs(1+f_k*f_j_other)));

                    if (f_k*f_j_other < -1.0f)
                        sign_k ^= 1;
                    }

                if (sign_k)
                    {
                    unsigned int *s_lookup;
                    unsigned int *s_sign;
                    unsigned int *s_len;
                    if (i_old)
                        {
                        s_lookup = &s_lookup_old[group*max_neighbors];
                        s_sign = &s_sign_old[group*max_neighbors];
                        s_len = &s_len_old[group];
                        }
                    else
                        {
                        s_lookup = &s_lookup_new[group*max_neighbors];
                        s_sign = &s_sign_new[group*max_neighbors];
                        s_len = &s_len_new[group];
                        }

                    // store sign change in table for neighbor k
                    unsigned int l = 0;
                    for (; l < *s_len; ++l)
                        {
                        if (s_lookup[l] == k)
                            break;
                        }

                    if (l == *s_len)
                        {
                        // add at the end
                        unsigned int insert_pos = *s_len;
                        if (insert_pos >= max_neighbors)
                            {
                            #if (__CUDA_ARCH__ >= 600)
                            atomicMax_system(&d_req_neighbors[i], insert_pos);
                            #else
                            atomicMax(&d_req_neighbors[i], insert_pos);
                            #endif
                            }

                        if (insert_pos < max_neighbors)
                            {
                            s_lookup[insert_pos] = k;
                            s_sign[insert_pos] = 1;
                            }

                        *s_len++;
                        }
                    else
                        {
                        // update sign for neighbor k
                        s_sign[l] ^= 1;
                        }
                    }
                } // end loop over terms

            // have we changed any neighbor's sign?
            unsigned int len = s_len_old[group];
            for (unsigned int l = 0; l < len; ++l)
                {
                if (s_sign_old[l])
                    atomicMax(&s_reject[group], 1);
                }
            // have we changed any neighbor's sign?
            len = s_len_new[group];
            for (unsigned int l = 0; l < len; ++l)
                {
                if (s_sign_new[l])
                    atomicMax(&s_reject[group], 1);
                }
            } // end depletants

        } // end if (active && move_active)

    __syncthreads();

    if (master && active)
        {
        d_reject_out[i] = s_reject[group];
        if (move_active)
            {
            // write out final free energy change
            d_F[i] = s_energy_old[group] - s_energy_new[group] + s_deltaF[group];
            }
        }
    }

//!< Kernel to evaluate convergence
__global__ void hpmc_accept(
                 const unsigned int *d_trial_move_type,
                 unsigned int *d_reject_in,
                 unsigned int *d_reject_out,
                 const Scalar *d_F,
                 unsigned int *d_condition,
                 const unsigned int seed,
                 const unsigned int select,
                 const unsigned int timestep,
                 const bool patch,
                 const bool have_auxiliary_variables,
                 const unsigned int nwork,
                 const unsigned work_offset)
    {
    // the particle we are handling
    unsigned int work_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (work_idx >= nwork)
        return;
    unsigned int i = work_idx + work_offset;

    // is this particle considered?
    bool move_active = d_trial_move_type[i] > 0;

    // combine with reject flag from gen_moves for particles which are always rejected
    bool reject = d_reject_out[i];

    if (move_active)
        {
        float deltaF = d_F[i]; // deltaF = F_old - F_new

        // Metropolis-Hastings
        hoomd::RandomGenerator rng_i(hoomd::RNGIdentifier::HPMCMonoAccept, seed, i, select, timestep);
        bool accept = !reject && ((!patch && !have_auxiliary_variables)
            || (hoomd::detail::generate_canonical<double>(rng_i) < slow::exp(deltaF)));

        if ((accept && d_reject_in[i]) || (!accept && !d_reject_in[i]))
            {
            // flag that we're not done yet (a trivial race condition upon write)
            *d_condition = 1;
            }

        // write out to device memory
        d_reject_out[i] = accept ? 0 : 1;
        }
    }

//! Generate number of depletants per particle
__global__ void generate_num_depletants(const unsigned int seed,
                                        const unsigned int timestep,
                                        const unsigned int select,
                                        const unsigned int num_types,
                                        const unsigned int depletant_type_a,
                                        const unsigned int depletant_type_b,
                                        const Index2D depletant_idx,
                                        const unsigned int work_offset,
                                        const unsigned int nwork,
                                        const Scalar *d_lambda,
                                        const Scalar4 *d_postype,
                                        unsigned int *d_n_depletants)
    {
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;

    if (idx >= nwork)
        return;

    idx += work_offset;

    hoomd::RandomGenerator rng_poisson(hoomd::RNGIdentifier::HPMCDepletantNum, idx, seed, timestep,
        select*depletant_idx.getNumElements() + depletant_idx(depletant_type_a,depletant_type_b));
    Index2D typpair_idx(num_types);
    unsigned int type_i = __scalar_as_int(d_postype[idx].w);
    d_n_depletants[idx] = hoomd::PoissonDistribution<Scalar>(
        d_lambda[type_i*depletant_idx.getNumElements()+depletant_idx(depletant_type_a,depletant_type_b)])(rng_poisson);
    }
} // end namespace kernel

//! Driver for kernel::hpmc_excell()
void hpmc_excell(unsigned int *d_excell_idx,
                 unsigned int *d_excell_size,
                 const Index2D& excli,
                 const unsigned int *d_cell_idx,
                 const unsigned int *d_cell_size,
                 const unsigned int *d_cell_adj,
                 const Index3D& ci,
                 const Index2D& cli,
                 const Index2D& cadji,
                 const unsigned int ngpu,
                 const unsigned int block_size)
    {
    assert(d_excell_idx);
    assert(d_excell_size);
    assert(d_cell_idx);
    assert(d_cell_size);
    assert(d_cell_adj);

    // determine the maximum block size and clamp the input block size down
    static int max_block_size = -1;
    if (max_block_size == -1)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_excell)));
        max_block_size = attr.maxThreadsPerBlock;
        }

    // setup the grid to run the kernel
    unsigned int run_block_size = min(block_size, (unsigned int)max_block_size);
    dim3 threads(run_block_size, 1, 1);
    dim3 grid(ci.getNumElements() / run_block_size + 1, 1, 1);

    hipLaunchKernelGGL(kernel::hpmc_excell, dim3(grid), dim3(threads), 0, 0, d_excell_idx,
                                           d_excell_size,
                                           excli,
                                           d_cell_idx,
                                           d_cell_size,
                                           d_cell_adj,
                                           ci,
                                           cli,
                                           cadji,
                                           ngpu);

    }

//! Kernel driver for kernel::hpmc_shift()
void hpmc_shift(Scalar4 *d_postype,
                int3 *d_image,
                const unsigned int N,
                const BoxDim& box,
                const Scalar3 shift,
                const unsigned int block_size)
    {
    assert(d_postype);
    assert(d_image);

    // setup the grid to run the kernel
    dim3 threads_shift(block_size, 1, 1);
    dim3 grid_shift(N / block_size + 1, 1, 1);

    hipLaunchKernelGGL(kernel::hpmc_shift, dim3(grid_shift), dim3(threads_shift), 0, 0, d_postype,
                                                      d_image,
                                                      N,
                                                      box,
                                                      shift);

    // after this kernel we return control of cuda managed memory to the host
    hipDeviceSynchronize();
    }


void hpmc_sum_energies(const unsigned int *d_update_order_by_ptl,
                 const unsigned int *d_trial_move_type,
                 const Scalar4 *d_trial_vel,
                 const unsigned int *d_reject_out_of_cell,
                 unsigned int *d_reject,
                 unsigned int *d_reject_out,
                 const unsigned int *d_nneigh,
                 const unsigned int *d_nlist,
                 const unsigned int N_old,
                 const unsigned int N,
                 const GPUPartition& gpu_partition,
                 const unsigned int maxn,
                 bool patch,
                 const unsigned int *d_nlist_patch_old,
                 const unsigned int *d_nlist_patch_new,
                 const unsigned int *d_nneigh_patch_old,
                 const unsigned int *d_nneigh_patch_new,
                 const float *d_energy_old,
                 const float *d_energy_new,
                 const unsigned int maxn_patch,
                 const unsigned int *d_deltaF_or_nneigh,
                 const unsigned int *d_deltaF_or_len,
                 const unsigned int *d_deltaF_or_nlist,
                 const float *d_deltaF_or_energy,
                 const Scalar *d_deltaF_or,
                 const unsigned int *d_deltaF_or_config,
                 const unsigned int maxn_deltaF_or,
                 const unsigned int *d_deltaF_nor_nneigh,
                 const unsigned int *d_deltaF_nor_len,
                 const unsigned int *d_deltaF_nor_k,
                 const unsigned int *d_deltaF_nor_nlist,
                 const float *d_deltaF_nor_energy,
                 const Scalar *d_deltaF_nor,
                 Scalar *d_F,
                 const unsigned int maxn_deltaF_nor,
                 const bool have_auxiliary_variables,
                 const unsigned int block_size,
                 const unsigned int tpp,
                 const unsigned int max_neighbors,
                 unsigned int *d_req_neighbors)
    {
    // determine the maximum block size and clamp the input block size down
    static int max_block_size = -1;
    if (max_block_size == -1)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_sum_energies)));
        max_block_size = attr.maxThreadsPerBlock;
        }

    // setup the grid to run the kernel
    unsigned int run_block_size = min(block_size, (unsigned int)max_block_size);

    // threads per particle
    unsigned int cur_tpp = min(run_block_size,tpp);
    while (run_block_size % cur_tpp != 0)
        cur_tpp--;

    unsigned int n_groups = run_block_size/cur_tpp;
    dim3 threads(cur_tpp, n_groups, 1);

    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;
        const unsigned int num_blocks = nwork/n_groups + 1;
        dim3 grid(num_blocks, 1, 1);

        unsigned int shared_bytes = n_groups * (3*sizeof(float) + 3*sizeof(unsigned int))
                                    + n_groups*max_neighbors*4*sizeof(unsigned int);

        hipLaunchKernelGGL(kernel::hpmc_sum_energies, grid, threads, shared_bytes, 0,
            d_update_order_by_ptl,
            d_trial_move_type,
            d_trial_vel,
            d_reject_out_of_cell,
            d_reject,
            d_reject_out,
            d_nneigh,
            d_nlist,
            N_old,
            N,
            nwork,
            range.first,
            maxn,
            patch,
            d_nlist_patch_old,
            d_nlist_patch_new,
            d_nneigh_patch_old,
            d_nneigh_patch_new,
            d_energy_old,
            d_energy_new,
            maxn_patch,
            d_deltaF_or_nneigh,
            d_deltaF_or_len,
            d_deltaF_or_nlist,
            d_deltaF_or_energy,
            d_deltaF_or,
            d_deltaF_or_config,
            maxn_deltaF_or,
            d_deltaF_nor_nneigh,
            d_deltaF_nor_len,
            d_deltaF_nor_k,
            d_deltaF_nor_nlist,
            d_deltaF_nor_energy,
            d_deltaF_nor,
            d_F,
            maxn_deltaF_nor,
            have_auxiliary_variables,
            max_neighbors,
            d_req_neighbors
            );
        }

    }

//!< Kernel to evaluate convergence
void hpmc_accept(const unsigned int *d_trial_move_type,
     unsigned int *d_reject_in,
     unsigned int *d_reject_out,
     const Scalar *d_F,
     unsigned int *d_condition,
     const unsigned int seed,
     const unsigned int select,
     const unsigned int timestep,
     const bool patch,
     const bool have_auxiliary_variables,
     const GPUPartition& gpu_partition,
     const unsigned int block_size)
    {
    // determine the maximum block size and clamp the input block size down
    static int max_block_size = -1;
    if (max_block_size == -1)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_accept)));
        max_block_size = attr.maxThreadsPerBlock;
        }

    // setup the grid to run the kernel
    unsigned int run_block_size = min(block_size, (unsigned int)max_block_size);

    dim3 threads(run_block_size, 1, 1);

    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;
        const unsigned int num_blocks = nwork/run_block_size + 1;
        dim3 grid(num_blocks, 1, 1);

        hipLaunchKernelGGL(kernel::hpmc_accept, grid, threads, 0, 0,
            d_trial_move_type,
            d_reject_in,
            d_reject_out,
            d_F,
            d_condition,
            seed,
            select,
            timestep,
            patch,
            have_auxiliary_variables,
            nwork,
            range.first);
        }
    }
} // end namespace gpu
} // end namespace hpmc

